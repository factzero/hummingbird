#include "hb_math_functions.h"


void hb_gpu_memcpy(const size_t N, const void *X, void *Y)
{
    if (X != Y)
    {
        CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));
    }
}